
#include <hip/hip_runtime.h>
#include <cstdio>
#define ARR_LEN 64

__global__ void reverse_arr(int *d_arr, int *d_out) {
    d_out[ARR_LEN - threadIdx.x - 1] = d_arr[threadIdx.x];
}

void print_arr(int *arr, int size){
    int i = 0;
    while (i < size) {
        std::printf("%d ", arr[i++]);
    }

    std::printf("\n");
}

void fill_arr(int *arr, int len) {
    int i;
    for (i=0; i<len; i++) {
        arr[i] = i;
    }
}

int main(void) {
    std::printf("Hello, world!");

    int *h_arr;
    int *d_arr, *d_out; 
    int size = ARR_LEN * sizeof(int);

    hipMalloc((void**) &d_arr, size);
    hipMalloc((void**) &d_out, size);
    h_arr = (int *)malloc(size);

    fill_arr(h_arr, ARR_LEN);

    hipMemcpy(d_arr, h_arr, size, hipMemcpyHostToDevice);
    std::printf("Reversing array\n");

    reverse_arr<<<1,ARR_LEN>>>(d_arr, d_out);

    hipMemcpy(h_arr, d_out, size, hipMemcpyDeviceToHost);
    print_arr(h_arr, ARR_LEN);

    hipFree(d_arr); hipFree(d_out);
    return 0;
}